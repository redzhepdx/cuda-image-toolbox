#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <math.h>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace cv;

// Sobel Filter Kernel function
__global__ void sobel_filter(unsigned char* d_img, unsigned char* d_out, const int ROWS, const int COLS) {
    int idx = threadIdx.x + blockIdx.x*blockDim.x;
    int idy = threadIdx.y + blockIdx.y*blockDim.y;

    if(idy > 0 && idy < ROWS - 1 && idx > 0 && idx < COLS - 1){
        int gx = d_img[(y-1)*COLS + (x-1)] + 2*d_img[(y)*COLS + (x-1)] +
                 d_img[(y+1)*COLS + (x-1)] - d_img[(y-1)*COLS + (x+1)] -
                 2*d_img[(y)*COLS + (x+1)] - d_img[(y+1)*COLS + (x+1)];

        int gy = d_img[(y-1)*COLS + (x-1)] + 2*d_img[(y-1)*COLS + (x)] +
                 d_img[(y-1)*COLS + (x+1)] - d_img[(y+1)*COLS + (x-1)] -
                 2*d_img[(y+1)*COLS + (x)] - d_img[(y+1)*COLS + (x+1)];

        int sum = abs(gx) + abs(gy);
        if(sum > 255) sum = 255;
        else if(sum < 0) sum = 0;

        d_out[idy*COLS + idx] = sum;
    }
}

// Apply sobel filter function
int applySobelFilter(Mat* in_img, Mat* out_img){
    // Create an empty image with same dimensions
    const int ROWS = in_img->rows;
    const int COLS = in_img->cols;

    // Copy the image info to an unsigned char array
    unsigned char* h_img = (unsigned char*)malloc(ROWS*COLS*sizeof(unsigned char));
    if(h_img == NULL){
        printf("ERROR: malloc - Couldn't allocate memory for the image channel input, on the host.");
        return -1;
    }
    for(int i = 0; i < ROWS; ++i)
        for(int j = 0; j < COLS; ++j)
            h_img[i*COLS + j] = in_img.at<uchar>(i, j);

    // Transfer the channel to the device
    unsigned char *d_img;
    checkCudaErrors(hipMalloc((void**)&d_img, ROWS*COLS*sizeof(unsigned char)));
    checkCudaErrors(hipMemcpy(d_img, h_img, ROWS*COLS*sizeof(unsigned char), hipMemcpyHostToDevice));

    // Determine the block and grid sizes
    dim3 blockSize(4, 4, 1);
    dim3 gridSize((int)((COLS - 1) / blockSize.x) + 1, (int)((ROWS - 1) / blockSize.y) + 1, 1);

    // Allocate memory on device for the image output
    unsigned char *d_out;
    checkCudaErrors(hipMalloc((void**)&d_out, ROWS*COLS*sizeof(unsigned char)));

    // Run the Sobel filter on the image
    sobel_filter<<<gridSize, blockSize>>>(d_img, d_out, ROWS, COLS);
    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

    // Transfer the channel back to host
    unsigned char* h_out = (unsigned char*)malloc(ROWS*COLS*sizeof(unsigned char));
    if(h_out == NULL){
        printf("ERROR: malloc - Couldn't allocate memory for the image channel output, on the host.");
        return -1;
    }
    checkCudaErrors(hipMemcpy(h_out, d_out, ROWS*COLS*sizeof(unsigned char), hipMemcpyDeviceToHost));

    // Save the channel to the Mat
    for(int i = 0; i < ROWS; ++i)
        for(int j = 0; j < COLS; ++j)
            out_img->at<uchar>(i, j) = h_out[i*COLS + j];

    // Cleanup
    if(h_img != NULL) free(h_img);
    if(h_out != NULL) free(h_out);
    checkCudaErrors(hipFree(d_img));
    checkCudaErrors(hipFree(d_out));
    return 0;
}

// Main function
int main() {
    // Load the image
    Mat in_img = imread("lana.jpg", CV_LOAD_IMAGE_GRAYSCALE);
    if(!in_img.data) {
        printf("ERROR: Couldn't open the image.");
        return -1;
    }

    // Display the input image
    imshow("Input Image", in_img);
    waitKey(0);

    // Create an empty image with same dimensions
    Mat out_img = Mat::zeros(in_img.rows, in_img.cols, CV_8UC1);

    // Apply the filter
    int result = applySobelFilter(&in_img, &out_img);

    // Display the output image
    if(result == 0){
        imshow("Output Image", out_img);
        waitKey(0);
    }

    return result;
}
